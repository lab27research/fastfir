#include "hip/hip_runtime.h"
#include "FastFirGPU1.h"

#include <algorithm>

FastFirGPU1::FastFirGPU1(float* mask, int mask_samps, int input_samps,
                         int buffers_per_call, bool contiguous)
    : FastFir(mask, mask_samps, input_samps, buffers_per_call, contiguous) {

    //Choose an FFT Size equal to next power of 2
    fft_size_ = FastFir::getFFTSize(mask_samps_, input_samps_);

    //Allocate device memory
    size_t io_buffer_bytes = sizeof(float) * 2 * buffers_per_call_ * fft_size_;
    size_t mask_buffer_bytes = sizeof(float) * 2 * fft_size_;
    DEVICE_MALLOC((void**)&d_io_buffer_, io_buffer_bytes);
    DEVICE_MALLOC((void**)&d_mask_buffer_, mask_buffer_bytes);

    //Initialize mask buffer
    size_t mask_bytes = sizeof(float) * 2 * mask_samps_;
    size_t non_mask_bytes = sizeof(float) * 2 * (fft_size_ - mask_samps_);
    checkCudaErrors(hipMemcpy(d_mask_buffer_, mask, mask_bytes, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemset(&d_mask_buffer_[2 * mask_samps_], 0, non_mask_bytes));

    hipfftHandle temp_plan;
    checkCudaErrors(hipfftCreate(&temp_plan));
    size_t workSize;
    checkCudaErrors(hipfftMakePlan1d(temp_plan, fft_size_, HIPFFT_C2C, 1, &workSize));
    checkCudaErrors(hipfftExecC2C(temp_plan, (hipfftComplex*)d_mask_buffer_, (hipfftComplex*)d_mask_buffer_, HIPFFT_FORWARD));
    checkCudaErrors(hipfftDestroy(temp_plan));

    //Initialize transfer streams
    checkCudaErrors(hipStreamCreate(&transfer1_stream_));
    checkCudaErrors(hipStreamCreate(&transfer2_stream_));

    //Default to buffers_per_call_, with a max of 8
    //Note: each one must allocate their own FFT working buffers!
    // todo: recommend checking GPU memory and warning/limiting here
    initProcStreams(std::min(8, buffers_per_call_));

    //Create one event per processing buffer
    transfer1_done_events_.resize(buffers_per_call_);
    kernels_done_events_.resize(buffers_per_call_);
    for (int ii = 0; ii < buffers_per_call_; ii++) {
        checkCudaErrors(hipEventCreate(&transfer1_done_events_[ii]));
        checkCudaErrors(hipEventCreate(&kernels_done_events_[ii]));
    }

    //Execute plans at least once to ensure no first-call overhead
    checkCudaErrors(hipMemset(d_io_buffer_, 0, io_buffer_bytes));
    for (int ii = 0; ii < std::min((int)fwd_plans_.size(), buffers_per_call_); ii++) {
        float* d_io_ptr = &d_io_buffer_[2 * ii * fft_size_];
        checkCudaErrors(hipfftExecC2C(fwd_plans_[ii], (hipfftComplex*)d_io_ptr, (hipfftComplex*)d_io_ptr, HIPFFT_FORWARD));
    }
}

FastFirGPU1::~FastFirGPU1() {
    //Destroy events
    for (int ii = 0; ii < buffers_per_call_; ii++) {
        checkCudaErrors(hipEventDestroy(transfer1_done_events_[ii]));
        checkCudaErrors(hipEventDestroy(kernels_done_events_[ii]));
    }

    //Destroy streams/plans
    initProcStreams(0);

    //Free device memory
    DEVICE_FREE(d_io_buffer_);
    DEVICE_FREE(d_mask_buffer_);

}

void FastFirGPU1::run(float* input, float* output) {
    size_t input_bytes = sizeof(float) * 2 * input_samps_;
    size_t non_input_bytes = sizeof(float) * 2 * (fft_size_ - input_samps_);
    int output_samps_0sided = FastFir::getOutputSampsNoTransient(mask_samps_, input_samps_);
    int output_samps_1sided = FastFir::getOutputSamps1Sided(mask_samps_, input_samps_);
    int output_samps_2sided = FastFir::getOutputSamps2Sided(mask_samps_, input_samps_);
    size_t output_bytes_0sided = sizeof(float) * 2 * output_samps_0sided;
    size_t output_bytes_1sided = sizeof(float) * 2 * output_samps_1sided;
    size_t output_bytes_2sided = sizeof(float) * 2 * output_samps_2sided;
    int left_transient_samps = output_samps_1sided - output_samps_0sided;

    ////Determine kernal parameters
    int tpb = getMaxThreadsPerBlock(0);
    //For kernels processing full fft size
    int num_blocks1 = getNumBlocks(tpb, fft_size_);
    //For kernels processing only transients
    int num_blocks2 = getNumBlocks(tpb, left_transient_samps);

    //Calculate fft scaling
    float scale = ((double)1.0) / fft_size_;

    //Output pointer movement depends on if we are using contiguous buffers
    float* h_output_ptr = output;

    int num_proc_streams = proc_streams_.size();
    for (int ii = 0; ii < buffers_per_call_; ii++) {
        int proc_stream_index = ii % num_proc_streams;

        //Choose streams
        hipStream_t stream1 = transfer1_stream_;
        hipStream_t stream2 = proc_streams_[proc_stream_index];
        hipStream_t stream3 = transfer2_stream_;

        //Choose cufft plans
        hipfftHandle fwd_plan = fwd_plans_[proc_stream_index];
        hipfftHandle rev_plan = rev_plans_[proc_stream_index];

        //Set buffer pointers
        float* d_io_ptr = &d_io_buffer_[2 * ii * fft_size_];
        float* h_input_ptr = &input[2 * ii * input_samps_];

        //Transfer1 : H->D : Move input samples to device and zero pad
        checkCudaErrors(hipMemcpyAsync(d_io_ptr, h_input_ptr, input_bytes, hipMemcpyHostToDevice, stream1));
        checkCudaErrors(hipMemsetAsync(&d_io_ptr[2 * input_samps_], 0, non_input_bytes));
        checkCudaErrors(hipEventRecord(transfer1_done_events_[ii], stream1));

        //Run fwd fft
        checkCudaErrors(hipStreamWaitEvent(stream2, transfer1_done_events_[ii]));
        checkCudaErrors(hipfftExecC2C(fwd_plan, (hipfftComplex*)d_io_ptr, (hipfftComplex*)d_io_ptr, HIPFFT_FORWARD));

        //Run cpx mpy/scaling kernel
        vectorCpxMpy << <num_blocks1, tpb, 0, stream2 >> > (d_io_ptr, d_mask_buffer_, d_io_ptr, fft_size_);
        checkCudaErrors(hipPeekAtLastError());

        vectorCpxScale << <num_blocks1, tpb, 0, stream2 >> > (d_io_ptr, d_io_ptr, scale, fft_size_);
        checkCudaErrors(hipPeekAtLastError());

        //Run rev fft
        checkCudaErrors(hipfftExecC2C(rev_plan, (hipfftComplex*)d_io_ptr, (hipfftComplex*)d_io_ptr, HIPFFT_BACKWARD));

        if (contiguous_) {
            //For contiguous, add in transient from previous kernels (need to wait until they are finished)
            // Note: for all buffers except the first
            if (ii != 0) {
                checkCudaErrors(hipStreamWaitEvent(stream2, kernels_done_events_[ii - 1]));
                float* prev_d_io_ptr = &d_io_buffer_[2 * (ii - 1) * fft_size_];
                vectorCpxAdd << <num_blocks2, tpb, 0, stream2 >> > (d_io_ptr, &prev_d_io_ptr[2 * output_samps_1sided], d_io_ptr, left_transient_samps);
                checkCudaErrors(hipPeekAtLastError());
            }
        }
        checkCudaErrors(hipEventRecord(kernels_done_events_[ii], stream2));


        //Transfer2 : D->H : Move output samples to host
        checkCudaErrors(hipStreamWaitEvent(stream3, kernels_done_events_[ii]));
        if (!contiguous_) {
            //Simply move data to its respective output buffer
            checkCudaErrors(hipMemcpyAsync(h_output_ptr, d_io_ptr, output_bytes_2sided, hipMemcpyDeviceToHost, stream3));

            h_output_ptr += 2 * output_samps_2sided;
        }
        else {
            //We need to add in overlaps
            if (ii != buffers_per_call_ - 1) {
                //Simply move the first data into output buffer
                checkCudaErrors(hipMemcpyAsync(h_output_ptr, d_io_ptr, output_bytes_1sided, hipMemcpyDeviceToHost, stream3));
                h_output_ptr += 2 * output_samps_1sided;
            }
            else {
                //Copy full 2-sided result for last buffer
                checkCudaErrors(hipMemcpyAsync(h_output_ptr, d_io_ptr, output_bytes_2sided, hipMemcpyDeviceToHost, stream3));
                h_output_ptr += 2 * output_samps_2sided;
            }

        }
    }

    //Synchronize all streams
    hipStreamSynchronize(transfer1_stream_);
    for (int ii = 0; ii < num_proc_streams; ii++) {
        hipStreamSynchronize(proc_streams_[ii]);
    }
    hipStreamSynchronize(transfer2_stream_);
}


//Allows override of number of streams
void FastFirGPU1::setNumProcStreams(int num_streams) {
    initProcStreams(num_streams);
}

void FastFirGPU1::initProcStreams(int num_streams) {
    //De-allocate any currently created strams/plans
    if (proc_streams_.size() != 0) {
        for (int ii = 0; ii < proc_streams_.size(); ii++) {
            checkCudaErrors(hipfftDestroy(fwd_plans_[ii]));
            checkCudaErrors(hipfftDestroy(rev_plans_[ii]));
            checkCudaErrors(hipStreamDestroy(proc_streams_[ii]));
        }
    }

    //Initialize new configuration
    fwd_plans_.resize(num_streams);
    rev_plans_.resize(num_streams);
    proc_streams_.resize(num_streams);
    for (int ii = 0; ii < num_streams; ii++) {
        //Initialize streams
        checkCudaErrors(hipStreamCreate(&proc_streams_[ii]));

        //Create cufft plans
        hipfftCreate(&fwd_plans_[ii]);
        checkCudaErrors(hipfftCreate(&rev_plans_[ii]));
        size_t workSize;
        checkCudaErrors(hipfftMakePlan1d(fwd_plans_[ii], fft_size_, HIPFFT_C2C, 1, &workSize));
        checkCudaErrors(hipfftMakePlan1d(rev_plans_[ii], fft_size_, HIPFFT_C2C, 1, &workSize));

        //Associate streams to plans
        checkCudaErrors(hipfftSetStream(fwd_plans_[ii], proc_streams_[ii]));
        checkCudaErrors(hipfftSetStream(rev_plans_[ii], proc_streams_[ii]));
    }
}

//Vectorized complex multiply
__global__ void vectorCpxMpy(float* input1, float* input2, float* output, int NN) {
    //One dimensional block configuration
    int ii = blockIdx.x * blockDim.x + threadIdx.x;

    if (ii < NN) {
        int offset = 2 * ii;
        float* ptr1 = input1 + offset;//input1 location for this thread
        float* ptr2 = input2 + offset;//input2 location for this thread
        float* ptr3 = output + offset;//output location for this thread

        float aa = *(ptr1);
        float bb = *(ptr1 + 1);
        float cc = *(ptr2);
        float dd = *(ptr2 + 1);

        *(ptr3) = aa * cc - bb * dd;
        *(ptr3 + 1) = aa * dd + bb * cc;
    }

}

__global__ void vectorCpxScale(float* input1, float* output, float scale, int NN) {
    //One dimensional block configuration
    int ii = blockIdx.x * blockDim.x + threadIdx.x;

    if (ii < NN) {
        int offset = 2 * ii;
        float* ptr1 = input1 + offset;//input1 location for this thread
        float* ptr2 = output + offset;//output location for this thread

        float aa = *(ptr1);
        float bb = *(ptr1 + 1);

        *(ptr2) = aa * scale;
        *(ptr2 + 1) = bb * scale;
    }
}

__global__ void vectorCpxAdd(float* input1, float* input2, float* output, int NN) {
    //One dimensional block configuration
    int ii = blockIdx.x * blockDim.x + threadIdx.x;

    if (ii < NN) {
        int offset = 2 * ii;
        float* ptr1 = input1 + offset;//input1 location for this thread
        float* ptr2 = input2 + offset;//input2 location for this thread
        float* ptr3 = output + offset;//output location for this thread

        float aa = *(ptr1);
        float bb = *(ptr1 + 1);
        float cc = *(ptr2);
        float dd = *(ptr2 + 1);

        *(ptr3) = aa + cc;
        *(ptr3 + 1) = bb + dd;
    }
}